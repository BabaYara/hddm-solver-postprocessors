#include "hip/hip_runtime.h"
#include "LinearBasis.h"
#include "Data.h"

#include <algorithm> // min & max

#define CAT(kernel, name) name##_kernel
#define KERNEL(name) CAT(kernel, name)

// CUDA 8.0 introduces sm_60_atomic_functions.h with atomicAdd(double*, double)
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
inline __attribute__((always_inline)) __device__ double atomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do
	{
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val + __longlong_as_double(assumed)));
	}
	while (assumed != old);

	return __longlong_as_double(old);
}
#endif // __CUDA_ARCH__

using namespace NAMESPACE;
using namespace std;

class Device;

__global__ void KERNEL(FUNCNAME)(
	const int dim, const int nno, const int DofPerNode, const int count, const double* const* x_,
	const int* nfreqs_, const XPS::Device* xps_, const int* szxps_, double** xpv_, const Chains::Device* chains_,
	const Matrix<double>::Device* surplus_, double** value_)
{
	for (int many = 0; many < COUNT; many++)
	{
		const double* x = x_[many];
		const int& nfreqs = nfreqs_[many];
		const XPS::Device& xps = xps_[many];
		double* xpv = xpv_[blockIdx.x];
		const Chains::Device& chains = chains_[many];
		const Matrix<double>::Device& surplus = surplus_[many];
		double* value = value_[many];

		// Loop to calculate all unique xp values.
		for (int i = threadIdx.x, e = szxps_[many]; i < e; i += blockDim.x)
		{
			const Index<uint16_t>& index = xps(i);
			const uint32_t& j = index.index;
			double xp = LinearBasis(x[j], index.i, index.j);
			xpv[i] = fmax(0.0, xp);
		}

		__syncthreads();

		// Loop to calculate scaled surplus product.
		for (int i = blockIdx.x; i < NNO; i += gridDim.x)
		{
			double temp = 1.0;
			for (int ifreq = 0; ifreq < nfreqs; ifreq++)
			{
				// Early exit for shorter chains.
				int32_t idx = chains(i * nfreqs + ifreq);
				if (!idx) break;

				temp *= xpv[idx];
				if (!temp) goto next;
			}

			for (int Dof_choice = threadIdx.x; Dof_choice < DOF_PER_NODE; Dof_choice += blockDim.x)
				atomicAdd(&value[Dof_choice], temp * surplus(i, Dof_choice));
		
		next :

			continue;
		}
	}
}

extern "C" void FUNCNAME(
	Device* device,
	const int dim, const int nno, const int DofPerNode, const int count, const double* const* x_,
	const int* nfreqs_, const XPS::Device* xps_, const int* szxps_, const Chains::Device* chains_,
	const Matrix<double>::Device* surplus_, double** value_)
{
	double** xDev = NULL;
	CUDA_ERR_CHECK(hipMalloc(&xDev, sizeof(double*) * count));
	vector<double*> x(count);
	Matrix<double>::Device xMatrixDev(count, dim);
	for (int i = 0; i < count; i++)
	{
		CUDA_ERR_CHECK(hipMemcpy(xMatrixDev.getData(i, 0), x_[i], sizeof(double) * dim,
			hipMemcpyHostToDevice));
		x[i] = xMatrixDev.getData(i, 0);
	}
	CUDA_ERR_CHECK(hipMemcpy(&xDev[0], &x[0], sizeof(double*) * count,
		hipMemcpyHostToDevice));

	double** valueDev = NULL;
	CUDA_ERR_CHECK(hipMalloc(&valueDev, sizeof(double*) * count));
	vector<double*> value(count);
	Matrix<double>::Device valueMatrixDev(count, dim);
	for (int i = 0; i < count; i++)
	{
		value[i] = valueMatrixDev.getData(i, 0);
		CUDA_ERR_CHECK(hipMemset(value[i], 0, sizeof(double) * dim));
	}
	CUDA_ERR_CHECK(hipMemcpy(&valueDev[0], &value[0], sizeof(double*) * count,
		hipMemcpyHostToDevice));

	int* szxpsDev = NULL;
	CUDA_ERR_CHECK(hipMalloc(&szxpsDev, sizeof(int) * count));
	CUDA_ERR_CHECK(hipMemcpy(&szxpsDev[0], &szxps_[0], sizeof(int) * count,
		hipMemcpyHostToDevice)); 

	// Choose CUDA compute grid.
	int szblock = 128;
	int nblocks = nno / szblock;
	if (nno % szblock) nblocks++;

	// Prepare the XPV buffer vector sized to max xps.size()
	// across all states.
	int szxpv = 0;
	for (int many = 0; many < count; many++)
		szxpv = max(szxpv, szxps_[many]);

	double** xpvDev = NULL;
	CUDA_ERR_CHECK(hipMalloc(&xpvDev, sizeof(double*) * nblocks));
	vector<double*> xpv(nblocks);
	Matrix<double>::Device xpvMatrixDev(nblocks, szxpv);
	for (int i = 0; i < nblocks; i++)
		xpv[i] = xpvMatrixDev.getData(i, 0);
	CUDA_ERR_CHECK(hipMemcpy(&xpvDev[0], &xpv[0], sizeof(double*) * nblocks,
		hipMemcpyHostToDevice));
	
	// Launch the kernel.
	KERNEL(FUNCNAME)<<<nblocks, szblock>>>(dim, nno, DofPerNode, count, xDev,
		nfreqs_, xps_, szxpsDev, xpvDev, chains_, surplus_, valueDev);

	CUDA_ERR_CHECK(hipDeviceSynchronize());
	
	for (int i = 0; i < count; i++)
		CUDA_ERR_CHECK(hipMemcpy(value_[i], valueMatrixDev.getData(i, 0), sizeof(double) * dim,
			hipMemcpyDeviceToHost));
	
	CUDA_ERR_CHECK(hipFree(xDev));
	CUDA_ERR_CHECK(hipFree(valueDev));
	CUDA_ERR_CHECK(hipFree(szxpsDev));
	CUDA_ERR_CHECK(hipFree(xpvDev));
}

