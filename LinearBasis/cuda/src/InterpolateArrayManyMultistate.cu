#include "hip/hip_runtime.h"
#define KERNEL_NAME InterpolateArrayManyMultistate_kernel_large_dim

#include "interpolator.h"
#include "Interpolate.h"
#include "LinearBasis.h"

#ifdef DEFERRED
#include <vector>
#endif

extern "C" __global__ void KERNEL_NAME(
#ifdef DEFERRED
	const X x_,
#else
 	const double* x_,
#endif
	const int dim, const int vdim, const int nno,
	const int Dof_choice_start, const int Dof_choice_end, const int count,
	const Matrix<int>::Device* index_, const Matrix<double>::Device* surplus_, double* value_)
{
	extern __shared__ double temps[];

	const int length = Dof_choice_end - Dof_choice_start + 1;

	for (int many = 0; many < COUNT; many++)
	{
		const Matrix<int>::Device& index = index_[many];
		const Matrix<double>::Device& surplus = surplus_[many];
		double* value = value_ + many * length;

		// The "i" is the index by nno, which could be either grid dimension X,
		// or partitioned between grid dimension X and block dimension Y.
		// In case of no partitioning, threadIdx.y is 0, and "i" falls back to
		// grid dimension X only.
		int i = blockIdx.x + threadIdx.y * blockDim.x;

		if (i >= nno) continue;

		// Each thread is assigned with a "j" loop index.
		// If DIM is larger than AVX_VECTOR_SIZE, each thread is
		// assigned with multiple "j" loop indexes.
		double temp = 1.0;
		#pragma no unroll
		for (int j = threadIdx.x; j < DIM; j += AVX_VECTOR_SIZE)
		{
			double xp = LinearBasis(x(j + many * DIM), index(i, j), index(i, j + vdim));
			temp *= max(0.0, xp);
		}
	
		// Multiply all partial temps within a warp.
		temp = warpReduceMultiply(temp);
	
		// Gather temps from all participating warps corresponding to the single DIM
		// into a shared memory array.
		int lane = threadIdx.x % warpSize;
		int warpId = threadIdx.x / warpSize;
		int nwarps = blockDim.x / warpSize;
		if (lane == 0)
			temps[warpId + threadIdx.y * nwarps] = temp;

		// Wait for all partial reductions.
		__syncthreads();

		// We can only exit at this point, when all threads in block are synchronized.
		if (!temp) continue;

		// Read from shared memory only if that warp existed.
		temp = (threadIdx.x < blockDim.x / warpSize) ? temps[lane + threadIdx.y * nwarps] : 1.0;

		// Final reduction within the first warp.
		if (warpId == 0)
		{
			temp = warpReduceMultiply(temp);

			// Store result into shared memory to broadcast across all warps.
			if (threadIdx.x == 0)
				temps[threadIdx.y * nwarps] = temp;
		}

		// Wait for the zero thread of the first warp to share temp value in shared memory.
		__syncthreads();

		// Load final reduction value from shared memory.
		temp = temps[threadIdx.y * nwarps];

		// Atomically add to the output value.
		// Uses double precision atomicAdd code above, since all
		// generations of GPUs before Pascal do not have double atomicAdd builtin.
		for (int j = Dof_choice_start + threadIdx.x; j <= Dof_choice_end; j += blockDim.x)
			atomicAdd(&value[j - Dof_choice_start], temp * surplus(i, j));
	}
}

extern "C" void FUNCNAME(
	Device* device,
	const int dim, const int nno,
	const int Dof_choice_start, const int Dof_choice_end, const int count, const double* const* x,
	const Matrix<int>::Device* index, const Matrix<double>::Device* surplus, double** value)
{
	// Configure kernel compute grid.
	int vdim = 1;
	dim3 blockDim(1, 1, 1);
	dim3 gridDim(1, 1, 1);
	int nwarps = 1;
	configureKernel(device, dim, nno, vdim, blockDim, gridDim, nwarps);

#ifdef DEFERRED
	std::vector<double> vx;
	vx.resize(DIM * COUNT);
	X* dx = (X*)&vx[0];
#else	
	double* dx;
	CUDA_ERR_CHECK(hipMalloc(&dx, sizeof(double) * DIM * COUNT));
#endif

	const int length = Dof_choice_end - Dof_choice_start + 1;
	double* dvalue;
	CUDA_ERR_CHECK(hipMalloc(&dvalue, sizeof(double) * length * COUNT));
	
	hipStream_t stream;
	CUDA_ERR_CHECK(hipStreamCreate(&stream));
	for (int i = 0; i < COUNT; i++)
	{
#ifdef DEFERRED
		memcpy(&vx[0] + i * DIM, x[i], sizeof(double) * DIM);
#else
		CUDA_ERR_CHECK(hipMemcpyAsync(dx + i * DIM, x[i], sizeof(double) * DIM, hipMemcpyHostToDevice, stream));
#endif
	}
	CUDA_ERR_CHECK(hipMemsetAsync(dvalue, 0, sizeof(double) * length * COUNT, stream));
	CUDA_ERR_CHECK(hipStreamSynchronize(stream));
	CUDA_ERR_CHECK(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(
		InterpolateArrayManyMultistate_kernel_large_dim), hipSharedMemBankSizeEightByte));
	InterpolateArrayManyMultistate_kernel_large_dim<<<gridDim, blockDim, nwarps * sizeof(double), stream>>>(
#ifdef DEFERRED
		*dx,
#else
		dx,
#endif
		dim, vdim, nno, Dof_choice_start, Dof_choice_end, COUNT,
		index, surplus, dvalue);
	CUDA_ERR_CHECK(hipGetLastError());
	CUDA_ERR_CHECK(hipStreamSynchronize(stream));
	CUDA_ERR_CHECK(hipDeviceSynchronize());
	for (int i = 0; i < COUNT; i++)
		CUDA_ERR_CHECK(hipMemcpyAsync(value[i], dvalue + i * length, sizeof(double) * length, hipMemcpyDeviceToHost, stream));
	CUDA_ERR_CHECK(hipStreamSynchronize(stream));
	CUDA_ERR_CHECK(hipStreamDestroy(stream));

#ifndef DEFERRED
	CUDA_ERR_CHECK(hipFree(dx));	
#endif
	CUDA_ERR_CHECK(hipFree(dvalue));
}

