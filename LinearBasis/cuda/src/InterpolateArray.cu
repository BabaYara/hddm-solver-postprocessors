#include "hip/hip_runtime.h"
#include "LinearBasis.h"
#include "Data.h"

#include <algorithm> // min & max

#define CAT(kernel, name) name##_kernel
#define KERNEL(name) CAT(kernel, name)

// CUDA 8.0 introduces sm_60_atomic_functions.h with atomicAdd(double*, double)
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
inline __attribute__((always_inline)) __device__ double atomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do
	{
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val + __longlong_as_double(assumed)));
	}
	while (assumed != old);

	return __longlong_as_double(old);
}
#endif // __CUDA_ARCH__

using namespace NAMESPACE;
using namespace std;

class Device;

__global__ void KERNEL(FUNCNAME)(
	const int dim, const int nno, const int nnoPerBlock, const int DofPerNode, const double* x,
	const int nfreqs, const XPS::Device* xps_, const int szxps, double** xpv_, const Chains::Device* chains_,
	const Matrix<double>::Device* surplus_, double* value)
{
	const XPS::Device& xps = *xps_;
	double* xpv = xpv_[blockIdx.x];
	const Chains::Device& chains = *chains_;
	const Matrix<double>::Device& surplus = *surplus_;

	// Loop to calculate all unique xp values.
	for (int i = threadIdx.x, e = szxps; i < e; i += blockDim.x)
	{
		const Index<uint16_t>& index = xps(i);
		const uint32_t& j = index.index;
		double xp = LinearBasis(x[j], index.i, index.j);
		xpv[i] = fmax(0.0, xp);
	}

	__syncthreads();

#define szcache 4
	// Each thread hosts a part of blockDim.x-shared register cache
	// to accumulate nnoPerBlock intermediate additions.
	// blockDim.x -sharing is done due to limited number of registers
	// available per thread.
	double cache[szcache];
	for (int i = 0; i < szcache; i++)
		cache[i] = 0;
#undef szcache

	// Loop to calculate scaled surplus product.
	for (int i = blockIdx.x * nnoPerBlock, e = min(i + nnoPerBlock, NNO); i < e; i++)
	{
		double temp = 1.0;
		for (int ifreq = 0; ifreq < nfreqs; ifreq++)
		{
			// Early exit for shorter chains.
			int32_t idx = chains(i * nfreqs + ifreq);
			if (!idx) break;

			temp *= xpv[idx];
			if (!temp) goto next;
		}

		for (int Dof_choice = threadIdx.x, icache = 0; Dof_choice < DOF_PER_NODE; Dof_choice += blockDim.x, icache++)
			cache[icache] += temp * surplus(i, Dof_choice);
	
	next :

		continue;
	}

	for (int Dof_choice = threadIdx.x, icache = 0; Dof_choice < DOF_PER_NODE; Dof_choice += blockDim.x, icache++)
		atomicAdd(&value[Dof_choice], cache[icache]);
}

namespace {

class InterpolateArray
{
	Vector<double>::Device xVectorDev;
	Vector<double>::Device valueVectorDev;
	Matrix<double>::Device xpvMatrixDev;

public :

	int dim;
	int DofPerNode;

	int szblock;
	int nnoPerBlock;
	int nblocks;

	double* xDev;
	double* valueDev;
	double** xpvDev;

	InterpolateArray(int dim, int nno, int DofPerNode, const int szxps) :
		dim(dim), DofPerNode(DofPerNode),
		szblock(128), nnoPerBlock(16), nblocks(nno / nnoPerBlock + (nno % nnoPerBlock ? 1 : 0)),
		xDev(NULL), xVectorDev(dim),
		valueDev(NULL), valueVectorDev(dim),
		xpvDev(NULL)

	{
		xDev = xVectorDev.getData();
		valueDev = valueVectorDev.getData();

		// Prepare the XPV buffer vector sized to max xps.size()
		// across all states. Individual buffer for each CUDA block.
		CUDA_ERR_CHECK(hipMalloc(&xpvDev, sizeof(double*) * nblocks));
		vector<double*> xpv(nblocks);
		xpvMatrixDev.resize(nblocks, szxps);
		for (int i = 0; i < nblocks; i++)
			xpv[i] = xpvMatrixDev.getData(i, 0);
		CUDA_ERR_CHECK(hipMemcpy(&xpvDev[0], &xpv[0], sizeof(double*) * nblocks,
			hipMemcpyHostToDevice));
	}

	void load(const double* x)
	{
		CUDA_ERR_CHECK(hipMemcpy(xVectorDev.getData(), x, sizeof(double) * dim,
			hipMemcpyHostToDevice));

		CUDA_ERR_CHECK(hipMemset(valueVectorDev.getData(), 0, sizeof(double) * DOF_PER_NODE));
	}

	void save(double* value)
	{
		CUDA_ERR_CHECK(hipMemcpy(value, valueVectorDev.getData(), sizeof(double) * DOF_PER_NODE,
			hipMemcpyDeviceToHost));
	}

	~InterpolateArray()
	{
		CUDA_ERR_CHECK(hipFree(xpvDev));
	}
};

unique_ptr<InterpolateArray> interp;

} // namespace

extern "C" void FUNCNAME(
	Device* device,
	const int dim, const int nno, const int DofPerNode, const double* x,
	const int nfreqs, const XPS::Device* xps_, const int szxps, const Chains::Device* chains_,
	const Matrix<double>::Device* surplus_, double* value)
{
	if (!interp.get())
		interp.reset(new InterpolateArray(dim, nno, DofPerNode, szxps));

	interp->load(x);

	// Launch the kernel.
	KERNEL(FUNCNAME)<<<interp->nblocks, interp->szblock>>>(
		dim, nno, interp->nnoPerBlock, DofPerNode, interp->xDev,
		nfreqs, xps_, szxps, interp->xpvDev, chains_, surplus_, interp->valueDev);

	CUDA_ERR_CHECK(hipDeviceSynchronize());

	interp->save(value);
}

