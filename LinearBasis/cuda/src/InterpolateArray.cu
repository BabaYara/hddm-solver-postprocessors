#include "hip/hip_runtime.h"
#include "interpolator.h"
#include "LinearBasis.h"

#include "Data.h"
#include "Device.h"

using namespace cuda;

// CUDA 8.0 introduces sm_60_atomic_functions.h with atomicAdd(double*, double)
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
inline __attribute__((always_inline)) __device__ double atomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do
	{
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val + __longlong_as_double(assumed)));
	}
	while (assumed != old);

	return __longlong_as_double(old);
}
#endif // __CUDA_ARCH__

inline __attribute__((always_inline)) __device__ double warpReduceMultiply(double val)
{
	for (int offset = warpSize / 2; offset > 0; offset /= 2)
		val *= __shfl_down(val, offset);
	return val;
}

static __global__ void InterpolateArray_kernel_large_dim(
	const int dim, const int vdim, const int nno,
	const int Dof_choice_start, const int Dof_choice_end, const double* x,
	const Matrix<int>::Device* index_, const Matrix<double>::Device* surplus_, double* value)
{
	extern __shared__ double temps[];

	const Matrix<int>::Device& index = *index_;
	const Matrix<double>::Device& surplus = *surplus_;

	// The "i" is the index by nno, which could be either grid dimension X,
	// or partitioned between grid dimension X and block dimension Y.
	// In case of no partitioning, threadIdx.y is 0, and "i" falls back to
	// grid dimension X only.
	int i = blockIdx.x + threadIdx.y * blockDim.x;

	// Each thread is assigned with a "j" loop index.
	// If DIM is larger than AVX_VECTOR_SIZE, each thread is
	// assigned with multiple "j" loop indexes.
	double temp = 1.0;
	for (int j = threadIdx.x; (j < DIM) && (i < nno); j += AVX_VECTOR_SIZE)
	{
		double xp = LinearBasis(x[j], index(i, j), index(i, j + vdim));
		temp *= max(0.0, xp);
	}
	
	// Multiply all partial temps within a warp.
	temp = warpReduceMultiply(temp);
	
	// Gather temps from all participating warps corresponding to the single DIM
	// into a shared memory array.
	int lane = threadIdx.x % warpSize;
	int warpId = threadIdx.x / warpSize;
	int nwarps = blockDim.x / warpSize;
	if (lane == 0)
		temps[warpId + threadIdx.y * nwarps] = temp;

	// Wait for all partial reductions.
	__syncthreads();

	// We can only exit at this point, when all threads in block are synchronized.
	if (!temp) return;
	if (i >= nno) return;
	
	// Read from shared memory only if that warp existed.
	temp = (threadIdx.x < blockDim.x / warpSize) ? temps[lane + threadIdx.y * nwarps] : 1.0;

	// Final reduction within the first warp.
	if (warpId != 0) return;
	temp = warpReduceMultiply(temp);
	
	if (threadIdx.x != 0) return;

	// Atomically add to the output value.
	// Uses double precision atomicAdd code above, since all
	// generations of GPUs before Pascal do not have double atomicAdd builtin.
	// TODO: precompute the sum of surplus rows
	for (int j = Dof_choice_start; j <= Dof_choice_end; j++)
		atomicAdd(&value[j - Dof_choice_start], temp * surplus(i, j));
}

extern "C" void FUNCNAME(
	Device* device,
	const int dim, const int nno,
	const int Dof_choice_start, const int Dof_choice_end, const double* x,
	const Matrix<int>::Device* index, const Matrix<double>::Device* surplus, double* value)
{
	// Index arrays shall be padded to AVX_VECTOR_SIZE-element
	// boundary to keep up the required alignment.
	int vdim = dim / AVX_VECTOR_SIZE;
	if (dim % AVX_VECTOR_SIZE) vdim++;
	vdim *= AVX_VECTOR_SIZE;
	
	// Choose efficient grid block dimensions.
	dim3 blockDim(1, 1, 1);
	dim3 gridDim(1, 1, 1);

	// If DIM is larger than the warp size, then pick up aligned dim
	// as the first dimension.
	if (DIM >= device->warpSize)
	{
		// If DIM is larger than AVX_VECTOR_SIZE, assign multiple
		// indexes per thread, with stepping.
		if (DIM > AVX_VECTOR_SIZE)
			blockDim.x = AVX_VECTOR_SIZE;
		else
		{
			blockDim.x = DIM;
			if (blockDim.x % device->warpSize)
				blockDim.x += device->warpSize - blockDim.x % device->warpSize;
		}

		// If the first dimension is still smaller than AVX_VECTOR_SIZE,
		// pick up a part of nno to get a close value.
		if (blockDim.x < AVX_VECTOR_SIZE)
		{
			blockDim.y = AVX_VECTOR_SIZE / blockDim.x;
			if (AVX_VECTOR_SIZE % blockDim.x)
				blockDim.y++;
			
			gridDim.x = nno / blockDim.y;
			if (nno % blockDim.y)
				gridDim.x++;
		}
		else
		{
			// Otherwise, whole nno goes as grid dimension.
			gridDim.x = nno;
		}
	}
	else
	{
		// Pick up a part of nno to have a block of at least
		// AVX_VECTOR_SIZE.
		blockDim.x = AVX_VECTOR_SIZE;
		
		// Set the rest of nno for grid dimension.
		gridDim.x = nno / blockDim.x;
		if (nno % blockDim.x)
			gridDim.x++;
	}

	// Calculate the number of warps in block.
	// It shall denote the size of shared memory used for
	// inter-warp step of temp value reduction.
	int nwarps = (blockDim.x * blockDim.y) / device->warpSize;

	double* dx;
	CUDA_ERR_CHECK(hipMalloc(&dx, sizeof(double) * DIM));

	const int length = Dof_choice_end - Dof_choice_start + 1;
	double* dvalue;
	CUDA_ERR_CHECK(hipMalloc(&dvalue, sizeof(double) * length));
	
	hipStream_t stream;
	CUDA_ERR_CHECK(hipStreamCreate(&stream));
	CUDA_ERR_CHECK(hipMemcpyAsync(dx, x, sizeof(double) * DIM, hipMemcpyHostToDevice, stream));
	CUDA_ERR_CHECK(hipMemsetAsync(dvalue, 0, sizeof(double) * length, stream));
	InterpolateArray_kernel_large_dim<<<gridDim, blockDim, nwarps, stream>>>(
		dim, vdim, nno, Dof_choice_start, Dof_choice_end, dx, index, surplus, dvalue);
	CUDA_ERR_CHECK(hipGetLastError());
	CUDA_ERR_CHECK(hipMemcpyAsync(value, dvalue, sizeof(double) * length, hipMemcpyDeviceToHost, stream));
	CUDA_ERR_CHECK(hipStreamSynchronize(stream));
	CUDA_ERR_CHECK(hipStreamDestroy(stream));

	CUDA_ERR_CHECK(hipFree(dx));	
	CUDA_ERR_CHECK(hipFree(dvalue));
}

