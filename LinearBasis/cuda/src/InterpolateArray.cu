#include "hip/hip_runtime.h"
#include "interpolator.h"
#include "LinearBasis.h"

#include "Data.h"
#include "Device.h"

using namespace cuda;

// CUDA 8.0 introduces sm_60_atomic_functions.h with atomicAdd(double*, double)
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
inline __attribute__((always_inline)) __device__ double atomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do
	{
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val + __longlong_as_double(assumed)));
	}
	while (assumed != old);

	return __longlong_as_double(old);
}
#endif // __CUDA_ARCH__

inline __attribute__((always_inline)) __device__ double warpReduceMultiply(double val)
{
	for (int offset = warpSize / 2; offset > 0; offset /= 2)
		val *= __shfl_down(val, offset);
	return val;
}

inline __attribute__((always_inline)) __device__ double warpAllReduceMultiply(double val)
{
	for (int offset = warpSize / 2; offset > 0; offset /= 2)
		val *= __shfl_xor(val, offset);
	return val;
}

#ifdef DEFERRED
// Define a structure-container, that shall be used to pass x vector as
// a single value via kernel argument. As an argument, it will be implicitly
// loaded into device constant memory.
struct X
{
	double values[DIM];
};

// In case of deferred compilation, X vector is loaded as a kernel argument
// for speed (saves on separate memcpy call). However, if used, brain-damaged CUDA
// compiler copies entire array into local memory (STL/LDL). In order to avoid this,
// we do not use the X kernel argument in the code directly, and instead read
// from 5th argument of PTX kernel representation. Note this is a fragile and
// compiler-specific hack.
inline __attribute__((always_inline))  __device__ double x(int j)
{
	double ret;
	asm(
		".reg .u64 ptr, i;\n\t"
		"mov.u64 ptr, InterpolateArray_kernel_large_dim_param_5;\n\t"
		"cvt.u64.u32 i, %1;\n\t"
		"mad.lo.u64 ptr, i, 8, ptr;\n\t"
		"ld.param.cs.f64 %0, [ptr];"  : "=d"(ret) : "r"(j));
	return ret;
}
#else
#define x(j) x_[j]
#endif

extern "C" __global__ void InterpolateArray_kernel_large_dim(
	const int dim, const int vdim, const int nno,
	const int Dof_choice_start, const int Dof_choice_end,
#ifdef DEFERRED
	const X x_,
#else
 	const double* x_,
#endif
	const Matrix<int>::Device* index_, const Matrix<double>::Device* surplus_, double* value)
{
	extern __shared__ double temps[];

	const Matrix<int>::Device& index = *index_;
	const Matrix<double>::Device& surplus = *surplus_;

	// The "i" is the index by nno, which could be either grid dimension X,
	// or partitioned between grid dimension X and block dimension Y.
	// In case of no partitioning, threadIdx.y is 0, and "i" falls back to
	// grid dimension X only.
	int i = blockIdx.x + threadIdx.y * blockDim.x;

	if (i >= nno) return;

	// Each thread is assigned with a "j" loop index.
	// If DIM is larger than AVX_VECTOR_SIZE, each thread is
	// assigned with multiple "j" loop indexes.
	double temp = 1.0;
	#pragma no unroll
	for (int j = threadIdx.x; j < DIM; j += AVX_VECTOR_SIZE)
	{
		double xp = LinearBasis(x(j), index(i, j), index(i, j + vdim));
		temp *= max(0.0, xp);
	}
	
	// Multiply all partial temps within a warp.
	temp = warpReduceMultiply(temp);
	
	// Gather temps from all participating warps corresponding to the single DIM
	// into a shared memory array.
	int lane = threadIdx.x % warpSize;
	int warpId = threadIdx.x / warpSize;
	int nwarps = blockDim.x / warpSize;
	if (lane == 0)
		temps[warpId + threadIdx.y * nwarps] = temp;

	// Wait for all partial reductions.
	__syncthreads();

	// We can only exit at this point, when all threads in block are synchronized.
	if (!temp) return;

	// Read from shared memory only if that warp existed.
	temp = (threadIdx.x < blockDim.x / warpSize) ? temps[lane + threadIdx.y * nwarps] : 1.0;

	// Final reduction within the first warp.
	if (warpId == 0)
	{
		temp = warpReduceMultiply(temp);

		// Store result into shared memory to broadcast across all warps.
		if (threadIdx.x == 0)
			temps[threadIdx.y * nwarps] = temp;
	}

	// Wait for the zero thread of the first warp to share temp value in shared memory.
	__syncthreads();

	// Load final reduction value from shared memory.
	temp = temps[threadIdx.y * nwarps];

	// Atomically add to the output value.
	// Uses double precision atomicAdd code above, since all
	// generations of GPUs before Pascal do not have double atomicAdd builtin.
	for (int j = Dof_choice_start + threadIdx.x; j <= Dof_choice_end; j += blockDim.x)
		atomicAdd(&value[j - Dof_choice_start], temp * surplus(i, j));
}

extern "C" void FUNCNAME(
	Device* device,
	const int dim, const int nno,
	const int Dof_choice_start, const int Dof_choice_end, const double* x,
	const Matrix<int>::Device* index, const Matrix<double>::Device* surplus, double* value)
{
	// Index arrays shall be padded to AVX_VECTOR_SIZE-element
	// boundary to keep up the required alignment.
	int vdim = dim / AVX_VECTOR_SIZE;
	if (dim % AVX_VECTOR_SIZE) vdim++;
	vdim *= AVX_VECTOR_SIZE;
	
	// Choose efficient grid block dimensions.
	dim3 blockDim(1, 1, 1);
	dim3 gridDim(1, 1, 1);

	// If DIM is larger than the warp size, then pick up aligned dim
	// as the first dimension.
	if (DIM >= device->warpSize)
	{
		// If DIM is larger than AVX_VECTOR_SIZE, assign multiple
		// indexes per thread, with stepping.
		if (DIM > AVX_VECTOR_SIZE)
			blockDim.x = AVX_VECTOR_SIZE;
		else
		{
			blockDim.x = DIM;
			if (blockDim.x % device->warpSize)
				blockDim.x += device->warpSize - blockDim.x % device->warpSize;
		}

		// If the first dimension is still smaller than AVX_VECTOR_SIZE,
		// pick up a part of nno to get a close value.
		if (blockDim.x < AVX_VECTOR_SIZE)
		{
			blockDim.y = AVX_VECTOR_SIZE / blockDim.x;
			if (AVX_VECTOR_SIZE % blockDim.x)
				blockDim.y++;
			
			gridDim.x = nno / blockDim.y;
			if (nno % blockDim.y)
				gridDim.x++;
		}
		else
		{
			// Otherwise, whole nno goes as grid dimension.
			gridDim.x = nno;
		}
	}
	else
	{
		// Pick up a part of nno to have a block of at least
		// AVX_VECTOR_SIZE.
		blockDim.x = AVX_VECTOR_SIZE;
		
		// Set the rest of nno for grid dimension.
		gridDim.x = nno / blockDim.x;
		if (nno % blockDim.x)
			gridDim.x++;
	}

	// Calculate the number of warps in block.
	// It shall denote the size of shared memory used for
	// inter-warp step of temp value reduction.
	int nwarps = (blockDim.x * blockDim.y) / device->warpSize;

#ifdef DEFERRED
	X* dx = (X*)x;
#else
	double* dx;
	CUDA_ERR_CHECK(hipMalloc(&dx, sizeof(double) * DIM));
#endif

	const int length = Dof_choice_end - Dof_choice_start + 1;
	double* dvalue;
	CUDA_ERR_CHECK(hipMalloc(&dvalue, sizeof(double) * length));
	
	hipStream_t stream;
	CUDA_ERR_CHECK(hipStreamCreate(&stream));
#ifndef DEFERRED
	CUDA_ERR_CHECK(hipMemcpyAsync(dx, x, sizeof(double) * DIM, hipMemcpyHostToDevice, stream));
#endif
	CUDA_ERR_CHECK(hipMemsetAsync(dvalue, 0, sizeof(double) * length, stream));
	CUDA_ERR_CHECK(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(
		InterpolateArray_kernel_large_dim), hipSharedMemBankSizeEightByte));
	InterpolateArray_kernel_large_dim<<<gridDim, blockDim, nwarps * sizeof(double), stream>>>(
		dim, vdim, nno, Dof_choice_start, Dof_choice_end,
#ifdef DEFERRED
		*dx,
#else
		dx,
#endif
		index, surplus, dvalue);
	CUDA_ERR_CHECK(hipGetLastError());
	CUDA_ERR_CHECK(hipMemcpyAsync(value, dvalue, sizeof(double) * length, hipMemcpyDeviceToHost, stream));
	CUDA_ERR_CHECK(hipStreamSynchronize(stream));
	CUDA_ERR_CHECK(hipStreamDestroy(stream));

#ifndef DEFERRED
	CUDA_ERR_CHECK(hipFree(dx));
#endif
	CUDA_ERR_CHECK(hipFree(dvalue));
}

