#include "Devices.h"
#include "interpolator.h"

#include <iostream>

using namespace cuda;
using namespace std;

Devices::Devices()
{
	MPI_Process* process;
	MPI_ERR_CHECK(MPI_Process_get(&process));

	// Check for available CUDA GPU(s)
	int ngpus = 0;
	hipError_t hipError_t = hipGetDeviceCount(&ngpus);
	if (hipError_t == hipErrorNoDevice)
	{
		if (process->isMaster())
			cout << hipGetErrorString(hipError_t) << endl;
		ngpus = 0;
	}
	else
		CUDA_ERR_CHECK(hipError_t);
	devices.resize(ngpus);

	// Find out the total number of GPU(s) available to
	// all participating hosts.
	int ngpus_total = ngpus;
	MPI_ERR_CHECK(MPI_Allreduce(MPI_IN_PLACE, &ngpus_total, 1, MPI_INT,
		MPI_SUM, MPI_COMM_WORLD));

	if (process->isMaster())
		cout << ngpus_total << " GPU(s) available" << endl;
	cout << endl;
	
	struct hipDeviceProp_t props;
	for (int igpu = 0; igpu < ngpus; igpu++)
	{
		CUDA_ERR_CHECK(hipGetDeviceProperties(&props, igpu));
		devices[igpu].warpSize = props.warpSize;
	}
}

int Devices::getCount()
{
	return devices.size();
}

const Device* Devices::getDevice(int index) const
{
	return &devices[index];
}

Device* Devices::tryAcquire()
{
	Device* device = NULL;

	for (int i = 0; i < devices.size(); i++)
	{
		#pragma omp critical
		{
			if (devices[i].available && !device)
			{
				devices[i].available = 0;
				device = &devices[i];
			}
		}
		
		if (device) break;
	}
	
	return device;
}

void Devices::release(Device* device)
{
	if (!device) return;

	#pragma omp atomic
	device->available++;
}

namespace cuda
{
	Devices devices;
}

extern "C" Device* tryAcquireDevice()
{
	return devices.tryAcquire();
}

extern "C" void releaseDevice(Device* device)
{
	devices.release(device);
}

