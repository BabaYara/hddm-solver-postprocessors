#include "Devices.h"
#include "interpolator.h"

#include <iostream>

using namespace cuda;
using namespace std;

Devices::Devices()
{
	MPI_Process* process;
	MPI_ERR_CHECK(MPI_Process_get(&process));

	// Check for available CUDA GPU(s)
	int ngpus = 0;
	hipError_t hipError_t = hipGetDeviceCount(&ngpus);
	if (hipError_t == hipErrorNoDevice)
	{
		if (process->isMaster())
			cout << hipGetErrorString(hipError_t) << endl;
		ngpus = 0;
	}
	else
		CUDA_ERR_CHECK(hipError_t);
	devices.resize(ngpus);

	// Find out the total number of GPU(s) available to
	// all participating hosts.
	int ngpus_total = ngpus;
	MPI_ERR_CHECK(MPI_Allreduce(MPI_IN_PLACE, &ngpus_total, 1, MPI_INT,
		MPI_SUM, MPI_COMM_WORLD));

	if (process->isMaster())
		cout << ngpus_total << " GPU(s) available" << endl;
	cout << endl;
	
	struct hipDeviceProp_t props;
	for (int igpu = 0; igpu < ngpus; igpu++)
	{
		CUDA_ERR_CHECK(hipGetDeviceProperties(&props, igpu));
		int id[2];
		id[0] = props.pciBusID;
		id[1] = props.pciDeviceID;
		devices[igpu].id = *(long long*)id;
		devices[igpu].warpSize = props.warpSize;
		devices[igpu].cc = props.major * 10 + props.minor;

		CUDA_ERR_CHECK(hipSetDevice(igpu));

		CUDA_ERR_CHECK(hipDeviceReset());

		// Call hipSetDeviceFlags() with the hipDeviceMapHost flag
		// in order for the hipHostMallocMapped flag to have effect
		// in hipHostAlloc mapped memory allocations.
		CUDA_ERR_CHECK(hipSetDeviceFlags(hipDeviceMapHost));
	}
	
	CUDA_ERR_CHECK(hipSetDevice(0));
}

int Devices::getCount()
{
	return devices.size();
}

const Device* Devices::getDevice(int index) const
{
	return &devices[index];
}

Device* Devices::tryAcquire()
{
	Device* device = NULL;

	for (int i = 0; i < devices.size(); i++)
	{
		#pragma omp critical
		{
			if (devices[i].available && !device)
			{
				devices[i].available = 0;
				device = &devices[i];
			}
		}
		
		if (device) break;
	}
	
	return device;
}

void Devices::release(Device* device)
{
	if (!device) return;

	#pragma omp atomic
	device->available++;
}

namespace cuda
{
	Devices devices;
}

extern "C" Device* tryAcquireDevice()
{
	return devices.tryAcquire();
}

extern "C" void releaseDevice(Device* device)
{
	devices.release(device);
}

