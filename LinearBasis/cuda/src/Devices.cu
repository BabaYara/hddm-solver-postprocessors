#include "Devices.h"
#include "interpolator.h"

#include <iostream>
#include <memory>
#include <mutex>

using namespace NAMESPACE;
using namespace std;

Devices::Devices()
{
	MPI_Process* process;
	MPI_ERR_CHECK(MPI_Process_get(&process));

	// Check for available CUDA GPU(s)
	int ngpus = 0;
	CUDA_ERR_CHECK(hipGetLastError());
	hipError_t hipError_t = hipGetDeviceCount(&ngpus);
	if (hipError_t == hipErrorNoDevice)
	{
		if (process->isMaster())
			cout << hipGetErrorString(hipError_t) << endl;
		ngpus = 0;
	}
	else
		CUDA_ERR_CHECK(hipError_t);
	devices.resize(ngpus);
	
	struct hipDeviceProp_t props;
	for (int igpu = 0; igpu < ngpus; igpu++)
	{
		CUDA_ERR_CHECK(hipGetDeviceProperties(&props, igpu));
		int id[2];
		id[0] = props.pciBusID;
		id[1] = props.pciDeviceID;
		devices[igpu].id = *(long long*)id;
		devices[igpu].warpSize = props.warpSize;
		devices[igpu].cc = props.major * 10 + props.minor;
	}
}

int Devices::getCount()
{
	return devices.size();
}

const Device* Devices::getDevice(int index) const
{
	return &devices[index];
}

Device* Devices::tryAcquire()
{
	Device* device = NULL;

	for (int i = 0; i < devices.size(); i++)
	{
		{
			static std::mutex mutex;
			std::lock_guard<std::mutex> lock(mutex);

			if (devices[i].available && !device)
			{
				devices[i].available = 0;
				device = &devices[i];
			}
		}
		
		if (device) break;
	}
	
	return device;
}

void Devices::release(Device* device)
{
	if (!device) return;

	{
		static std::mutex mutex;
		std::lock_guard<std::mutex> lock(mutex);

		device->available++;
	}
}

namespace NAMESPACE
{
	unique_ptr<Devices> devices;
}

extern "C" Device* tryAcquireDevice()
{
	if (!devices)
		devices.reset(new Devices());

	return devices->tryAcquire();
}

extern "C" void releaseDevice(Device* device)
{
	if (!devices)
		devices.reset(new Devices());

	devices->release(device);
}

